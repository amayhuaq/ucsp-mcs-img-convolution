#include "hip/hip_runtime.h"
#include <iostream>
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <>

#include "ImageHelper.h"

using namespace std;

__global__ void applySobelFilter(uchar *image, float *filter, uchar *out_image, int w, int h)
{
	int left, right, top, bottom, lefttop, righttop, leftbot, rightbot;

	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int offset = x + y * blockDim.x * gridDim.x;

	left = offset - 1;
	right = offset + 1;
	if (x == 0) left++;
	if (x == w - 1) right--;
	top = offset - w;
	bottom = offset + w;
	if (y == 0) top += w;
	if (y == w - 1) bottom -= w;

	lefttop = top - 1;
	righttop = top + 1;
	leftbot = bottom - 1;
	rightbot = bottom + 1;

	if (x == 0)	{
		lefttop = top;
		leftbot = bottom;
	}
	if (x == w - 1) {
		righttop = top;
		rightbot = bottom;
	}
	if (y == 0) {
		lefttop = top;
		righttop = top;
	}
	if (y == h - 1) {
		leftbot = bottom;
		rightbot = bottom;
	}

	int sum_x = (image[lefttop] * filter[0] + image[top] * filter[1] + image[righttop] * filter[2])
			+ (image[left] * filter[3] + image[offset] * filter[4] + image[right] * filter[5])
			+ (image[leftbot] * filter[6] + image[bottom] * filter[7] + image[rightbot] * filter[8]);

	int sum_y = (image[lefttop] * filter[0] + image[top] * filter[3] + image[righttop] * filter[6])
			+ (image[left] * filter[1] + image[offset] * filter[4] + image[right] * filter[7])
			+ (image[leftbot] * filter[2] + image[bottom] * filter[5] + image[rightbot] * filter[8]);

	out_image[offset] = sqrtf((sum_x * sum_x) + (sum_y * sum_y));
}

__device__ int getOffsetPos(int w, int h, int i, int j)
{
	if (i < 0 || i >= h || j < 0 || j >= w)
		return -1;
	return i * w + j;
}

__device__ int getRowFromPos(int pos, int w)
{
	return pos / w;
}

__device__ int getColFromPos(int pos, int w)
{
	return pos % w;
}

__global__ void applyFilter2(uchar *image, float *filter, uchar *out_image, int w, int h, int wf, int hf)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int offset = x + y * blockDim.x * gridDim.x;

	int i = getRowFromPos(offset, w);
	int j = getColFromPos(offset, w);

	if(i >= h || j >= w)
		return;

	float sumVals = 0;
	int cini, cfin, rini, rfin;

	cini = j - wf / 2;
	cfin = j + wf / 2;
	rini = i - hf / 2;
	rfin = i + hf / 2;
	cini = (cini < 0) ? 0 : cini;
	rini = (rini < 0) ? 0 : rini;
	cfin = (cfin >= w) ? w-1 : cfin;
	rfin = (rfin >= h) ? h-1 : rfin;

	for(int fi = 0; rini <= rfin && fi < hf; rini++, fi++) {
		for(int tj = cini, fj = 0; tj <= cfin && fj < wf; tj++, fj++) {
			sumVals += image[getOffsetPos(w, h, rini, tj)] * filter[getOffsetPos(wf, hf, fi, fj)];
		}
	}
	out_image[offset] = sumVals;
}

Mat applyFilter(Mat img, Filter *filter)
{
	int width, height;
	uchar *bitmap, *out_bitmap;
	uchar *dev_bitmap, *dev_out_bitmap;
	float *dev_filter;

	width = img.cols;
	height = img.rows;
	bitmap = convertMatToArray(img);
	out_bitmap = new uchar[height * width];

	cout << "Applying Filter: " << filter->h << " x " << filter->w << endl;
	cout << "Size image: " << width << " x " << height << endl;

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	hipMalloc((void**)&dev_filter, filter->w * filter->h * sizeof(float));
	hipMalloc((void**)&dev_bitmap, width * height * sizeof(uchar));
	hipMalloc((void**)&dev_out_bitmap, width * height * sizeof(uchar));

	hipMemcpy(dev_filter, filter->values, filter->w * filter->h * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_bitmap, bitmap, width * height * sizeof(uchar), hipMemcpyHostToDevice);

	int blocks = (width * height + MAX_THREADS - 1) / MAX_THREADS;
	int threads = MAX_THREADS;

	clock_t time = clock();
	switch (filter->type) {
		case SOBEL_FILTER:
			applySobelFilter << <blocks, threads >> >(dev_bitmap, dev_filter, dev_out_bitmap, width, height);
			break;
		default:
			applyFilter2 << <blocks, threads >> >(dev_bitmap, dev_filter, dev_out_bitmap, width, height, filter->w, filter->h);
			break;
	}
	hipMemcpy(out_bitmap, dev_out_bitmap, width * height * sizeof(uchar), hipMemcpyDeviceToHost);

	hipFree(dev_filter);
	hipFree(dev_out_bitmap);
	hipFree(dev_bitmap);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);
	cout << "Elapsed time: " << elapsedTime / 1000.0 << " secs\n";

	return convertArrayToMat(out_bitmap, width, height);
}

int main(int argc, char **argv)
{
	Mat img = loadImage("data/persona04.jpg", GREY_MODE);
	//Mat out = applyFilter(img, createFilter(SOBEL_FILTER, 3));
	Mat out = applyFilter(img, createFilter(AVG_FILTER, 21));

	showImage(img, "Input image");
	showImage(out, "Filtered image");

	return 0;
}
